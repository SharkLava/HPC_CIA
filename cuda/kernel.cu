
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned char uchar;

__host__ void errorexit(const char *s) {
  printf("\n%s\n", s);
  exit(EXIT_FAILURE);
}

__constant__ int Height;
__constant__ int Width;

__global__ void bicubic(uchar *bpixels, uchar *pixels) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  extern __shared__ int suma[][3];

  if (row >= Height || col >= Width)
    return;

  int z = blockIdx.z;
  int index = z * Height * Width + row * Width + col;
  if (row == 0 || col == 0 || row == Height - 1 || col == Width - 1)
    bpixels[index] =
        pixels[z * Height * Width / 4 + row / 2 * Width / 2 + col / 2];
  else {
    int sindex = +threadIdx.x * blockDim.y + threadIdx.y;
    suma[sindex][threadIdx.z] =
        pixels[z * Height * Width / 4 + (row - 1) / 2 * Width / 2 +
               (col + threadIdx.z - 1) / 2] +
        pixels[z * Height * Width / 4 + row / 2 * Width / 2 +
               (col + threadIdx.z - 1) / 2] +
        pixels[z * Height * Width / 4 + (row + 1) / 2 * Width / 2 +
               (col + threadIdx.z - 1) / 2];

    __syncthreads();
    if (threadIdx.z == 0) {
      int s = 0;
      for (int i = 0; i < 3; i++) {
        s += suma[sindex][i];
      }
      bpixels[index] = (uchar)(s / 9);
    }
  }
}

void cudaFunction(uchar *hnpixels, uchar *hpixels, int height, int width) {

  int n = 2;

  int new_height = n * height;
  int new_width = n * width;

  long memory = height * width * 3 * sizeof(uchar);
  long memory2 = new_height * new_width * 3 * sizeof(uchar);

  uchar *dpixels;
  uchar *dbpixels;

  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(Height), &new_height, sizeof(int), 0,
                                        hipMemcpyHostToDevice)) {
    errorexit("Error copying `Height` on the GPU");
  }
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(Width), &new_width, sizeof(int), 0,
                                        hipMemcpyHostToDevice)) {
    errorexit("Error copying `Width` on the GPU");
  }
  if (hipSuccess != hipMalloc((void **)&dpixels, memory)) {
    errorexit("Error allocating `dpixels` memory on the GPU");
  }
  if (hipSuccess != hipMalloc((void **)&dbpixels, memory2)) {
    errorexit("Error allocating `dbpixels` memory on the GPU");
  }
  if (hipSuccess !=
      hipMemcpy(dpixels, hpixels, memory, hipMemcpyHostToDevice)) {
    errorexit("Error copying results from host to device");
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int nn = 8;
  dim3 blockSize(nn, nn, 3);
  dim3 gridSize((new_height - 1) / blockSize.x + 1,
                (new_width - 1) / blockSize.y + 1, 3);

  hipEventRecord(start);
  bicubic<<<gridSize, blockSize, nn * nn * 3 * sizeof(int)>>>(dbpixels,
                                                              dpixels);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  if (hipSuccess !=
      hipMemcpy(hnpixels, dbpixels, memory2, hipMemcpyDeviceToHost)) {
    errorexit("Error copying results from device to host");
  }
  hipFree(dpixels);
  hipFree(dbpixels);
  printf("Time taken: %fms\n", milliseconds);
}
